#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include <opencv2/gpu/device/common.hpp>
#include <opencv2/gpu/device/vec_traits.hpp>
#include <opencv2/gpu/device/vec_math.hpp>
#include <opencv2/gpu/device/limits.hpp>

namespace cv { namespace gpu {
    namespace device
    {
        template <typename D>
        __global__ void Bayer2BGR_8u(const PtrStepb src, PtrStepSz<D> dst, const bool blue_last, const bool start_with_green)
        {
            const int s_x = blockIdx.x * blockDim.x + threadIdx.x;
            int s_y = blockIdx.y * blockDim.y + threadIdx.y;

            if (s_y >= dst.rows || (s_x << 2) >= dst.cols)
                return;

            s_y = ::min(::max(s_y, 1), dst.rows - 2);

            uchar4 patch[3][3];
            patch[0][1] = ((const uchar4*) src.ptr(s_y - 1))[s_x];
            patch[0][0] = ((const uchar4*) src.ptr(s_y - 1))[::max(s_x - 1, 0)];
            patch[0][2] = ((const uchar4*) src.ptr(s_y - 1))[::min(s_x + 1, ((dst.cols + 3) >> 2) - 1)];

            patch[1][1] = ((const uchar4*) src.ptr(s_y))[s_x];
            patch[1][0] = ((const uchar4*) src.ptr(s_y))[::max(s_x - 1, 0)];
            patch[1][2] = ((const uchar4*) src.ptr(s_y))[::min(s_x + 1, ((dst.cols + 3) >> 2) - 1)];

            patch[2][1] = ((const uchar4*) src.ptr(s_y + 1))[s_x];
            patch[2][0] = ((const uchar4*) src.ptr(s_y + 1))[::max(s_x - 1, 0)];
            patch[2][2] = ((const uchar4*) src.ptr(s_y + 1))[::min(s_x + 1, ((dst.cols + 3) >> 2) - 1)];

            D res0 = VecTraits<D>::all(numeric_limits<uchar>::max());
            D res1 = VecTraits<D>::all(numeric_limits<uchar>::max());
            D res2 = VecTraits<D>::all(numeric_limits<uchar>::max());
            D res3 = VecTraits<D>::all(numeric_limits<uchar>::max());

            if ((s_y & 1) ^ start_with_green)
            {
                const int t0 = (patch[0][1].x + patch[2][1].x + 1) >> 1;
                const int t1 = (patch[1][0].w + patch[1][1].y + 1) >> 1;

                const int t2 = (patch[0][1].x + patch[0][1].z + patch[2][1].x + patch[2][1].z + 2) >> 2;
                const int t3 = (patch[0][1].y + patch[1][1].x + patch[1][1].z + patch[2][1].y + 2) >> 2;

                const int t4 = (patch[0][1].z + patch[2][1].z + 1) >> 1;
                const int t5 = (patch[1][1].y + patch[1][1].w + 1) >> 1;

                const int t6 = (patch[0][1].z + patch[0][2].x + patch[2][1].z + patch[2][2].x + 2) >> 2;
                const int t7 = (patch[0][1].w + patch[1][1].z + patch[1][2].x + patch[2][1].w + 2) >> 2;

                if ((s_y & 1) ^ blue_last)
                {
                    res0.x = t1;
                    res0.y = patch[1][1].x;
                    res0.z = t0;

                    res1.x = patch[1][1].y;
                    res1.y = t3;
                    res1.z = t2;

                    res2.x = t5;
                    res2.y = patch[1][1].z;
                    res2.z = t4;

                    res3.x = patch[1][1].w;
                    res3.y = t7;
                    res3.z = t6;
                }
                else
                {
                    res0.x = t0;
                    res0.y = patch[1][1].x;
                    res0.z = t1;

                    res1.x = t2;
                    res1.y = t3;
                    res1.z = patch[1][1].y;

                    res2.x = t4;
                    res2.y = patch[1][1].z;
                    res2.z = t5;

                    res3.x = t6;
                    res3.y = t7;
                    res3.z = patch[1][1].w;
                }
            }
            else
            {
                const int t0 = (patch[0][0].w + patch[0][1].y + patch[2][0].w + patch[2][1].y + 2) >> 2;
                const int t1 = (patch[0][1].x + patch[1][0].w + patch[1][1].y + patch[2][1].x + 2) >> 2;

                const int t2 = (patch[0][1].y + patch[2][1].y + 1) >> 1;
                const int t3 = (patch[1][1].x + patch[1][1].z + 1) >> 1;

                const int t4 = (patch[0][1].y + patch[0][1].w + patch[2][1].y + patch[2][1].w + 2) >> 2;
                const int t5 = (patch[0][1].z + patch[1][1].y + patch[1][1].w + patch[2][1].z + 2) >> 2;

                const int t6 = (patch[0][1].w + patch[2][1].w + 1) >> 1;
                const int t7 = (patch[1][1].z + patch[1][2].x + 1) >> 1;

                if ((s_y & 1) ^ blue_last)
                {
                    res0.x = patch[1][1].x;
                    res0.y = t1;
                    res0.z = t0;

                    res1.x = t3;
                    res1.y = patch[1][1].y;
                    res1.z = t2;

                    res2.x = patch[1][1].z;
                    res2.y = t5;
                    res2.z = t4;

                    res3.x = t7;
                    res3.y = patch[1][1].w;
                    res3.z = t6;
                }
                else
                {
                    res0.x = t0;
                    res0.y = t1;
                    res0.z = patch[1][1].x;

                    res1.x = t2;
                    res1.y = patch[1][1].y;
                    res1.z = t3;

                    res2.x = t4;
                    res2.y = t5;
                    res2.z = patch[1][1].z;

                    res3.x = t6;
                    res3.y = patch[1][1].w;
                    res3.z = t7;
                }
            }

            const int d_x = (blockIdx.x * blockDim.x + threadIdx.x) << 2;
            const int d_y = blockIdx.y * blockDim.y + threadIdx.y;

            dst(d_y, d_x) = res0;
            if (d_x + 1 < dst.cols)
                dst(d_y, d_x + 1) = res1;
            if (d_x + 2 < dst.cols)
                dst(d_y, d_x + 2) = res2;
            if (d_x + 3 < dst.cols)
                dst(d_y, d_x + 3) = res3;
        }

        template <typename D>
        __global__ void Bayer2BGR_16u(const PtrStepb src, PtrStepSz<D> dst, const bool blue_last, const bool start_with_green)
        {
            const int s_x = blockIdx.x * blockDim.x + threadIdx.x;
            int s_y = blockIdx.y * blockDim.y + threadIdx.y;

            if (s_y >= dst.rows || (s_x << 1) >= dst.cols)
                return;

            s_y = ::min(::max(s_y, 1), dst.rows - 2);

            ushort2 patch[3][3];
            patch[0][1] = ((const ushort2*) src.ptr(s_y - 1))[s_x];
            patch[0][0] = ((const ushort2*) src.ptr(s_y - 1))[::max(s_x - 1, 0)];
            patch[0][2] = ((const ushort2*) src.ptr(s_y - 1))[::min(s_x + 1, ((dst.cols + 1) >> 1) - 1)];

            patch[1][1] = ((const ushort2*) src.ptr(s_y))[s_x];
            patch[1][0] = ((const ushort2*) src.ptr(s_y))[::max(s_x - 1, 0)];
            patch[1][2] = ((const ushort2*) src.ptr(s_y))[::min(s_x + 1, ((dst.cols + 1) >> 1) - 1)];

            patch[2][1] = ((const ushort2*) src.ptr(s_y + 1))[s_x];
            patch[2][0] = ((const ushort2*) src.ptr(s_y + 1))[::max(s_x - 1, 0)];
            patch[2][2] = ((const ushort2*) src.ptr(s_y + 1))[::min(s_x + 1, ((dst.cols + 1) >> 1) - 1)];

            D res0 = VecTraits<D>::all(numeric_limits<ushort>::max());
            D res1 = VecTraits<D>::all(numeric_limits<ushort>::max());

            if ((s_y & 1) ^ start_with_green)
            {
                const int t0 = (patch[0][1].x + patch[2][1].x + 1) >> 1;
                const int t1 = (patch[1][0].y + patch[1][1].y + 1) >> 1;

                const int t2 = (patch[0][1].x + patch[0][2].x + patch[2][1].x + patch[2][2].x + 2) >> 2;
                const int t3 = (patch[0][1].y + patch[1][1].x + patch[1][2].x + patch[2][1].y + 2) >> 2;

                if ((s_y & 1) ^ blue_last)
                {
                    res0.x = t1;
                    res0.y = patch[1][1].x;
                    res0.z = t0;

                    res1.x = patch[1][1].y;
                    res1.y = t3;
                    res1.z = t2;
                }
                else
                {
                    res0.x = t0;
                    res0.y = patch[1][1].x;
                    res0.z = t1;

                    res1.x = t2;
                    res1.y = t3;
                    res1.z = patch[1][1].y;
                }
            }
            else
            {
                const int t0 = (patch[0][0].y + patch[0][1].y + patch[2][0].y + patch[2][1].y + 2) >> 2;
                const int t1 = (patch[0][1].x + patch[1][0].y + patch[1][1].y + patch[2][1].x + 2) >> 2;

                const int t2 = (patch[0][1].y + patch[2][1].y + 1) >> 1;
                const int t3 = (patch[1][1].x + patch[1][2].x + 1) >> 1;

                if ((s_y & 1) ^ blue_last)
                {
                    res0.x = patch[1][1].x;
                    res0.y = t1;
                    res0.z = t0;

                    res1.x = t3;
                    res1.y = patch[1][1].y;
                    res1.z = t2;
                }
                else
                {
                    res0.x = t0;
                    res0.y = t1;
                    res0.z = patch[1][1].x;

                    res1.x = t2;
                    res1.y = patch[1][1].y;
                    res1.z = t3;
                }
            }

            const int d_x = (blockIdx.x * blockDim.x + threadIdx.x) << 1;
            const int d_y = blockIdx.y * blockDim.y + threadIdx.y;

            dst(d_y, d_x) = res0;
            if (d_x + 1 < dst.cols)
                dst(d_y, d_x + 1) = res1;
        }

        template <int cn>
        void Bayer2BGR_8u_gpu(PtrStepSzb src, PtrStepSzb dst, bool blue_last, bool start_with_green, hipStream_t stream)
        {
            typedef typename TypeVec<uchar, cn>::vec_type dst_t;

            const dim3 block(32, 8);
            const dim3 grid(divUp(dst.cols, 4 * block.x), divUp(dst.rows, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(Bayer2BGR_8u<dst_t>), hipFuncCachePreferL1) );

            Bayer2BGR_8u<dst_t><<<grid, block, 0, stream>>>(src, (PtrStepSz<dst_t>)dst, blue_last, start_with_green);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }
        template <int cn>
        void Bayer2BGR_16u_gpu(PtrStepSzb src, PtrStepSzb dst, bool blue_last, bool start_with_green, hipStream_t stream)
        {
            typedef typename TypeVec<ushort, cn>::vec_type dst_t;

            const dim3 block(32, 8);
            const dim3 grid(divUp(dst.cols, 2 * block.x), divUp(dst.rows, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(Bayer2BGR_16u<dst_t>), hipFuncCachePreferL1) );

            Bayer2BGR_16u<dst_t><<<grid, block, 0, stream>>>(src, (PtrStepSz<dst_t>)dst, blue_last, start_with_green);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template void Bayer2BGR_8u_gpu<3>(PtrStepSzb src, PtrStepSzb dst, bool blue_last, bool start_with_green, hipStream_t stream);
        template void Bayer2BGR_8u_gpu<4>(PtrStepSzb src, PtrStepSzb dst, bool blue_last, bool start_with_green, hipStream_t stream);
        template void Bayer2BGR_16u_gpu<3>(PtrStepSzb src, PtrStepSzb dst, bool blue_last, bool start_with_green, hipStream_t stream);
        template void Bayer2BGR_16u_gpu<4>(PtrStepSzb src, PtrStepSzb dst, bool blue_last, bool start_with_green, hipStream_t stream);
    }
}}

#endif /* CUDA_DISABLER */