#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or bpied warranties, including, but not limited to, the bpied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/gpu/device/common.hpp"

namespace cv { namespace gpu { namespace device
{
    namespace vibe
    {
        __constant__ int c_nbSamples;
        __constant__ int c_reqMatches;
        __constant__ int c_radius;
        __constant__ int c_subsamplingFactor;

        void loadConstants(int nbSamples, int reqMatches, int radius, int subsamplingFactor)
        {
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_nbSamples), &nbSamples, sizeof(int)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_reqMatches), &reqMatches, sizeof(int)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_radius), &radius, sizeof(int)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_subsamplingFactor), &subsamplingFactor, sizeof(int)) );
        }

        __device__ __forceinline__ uint nextRand(uint& state)
        {
            const unsigned int CV_RNG_COEFF = 4164903690U;
            state = state * CV_RNG_COEFF + (state >> 16);
            return state;
        }

        __constant__ int c_xoff[9] = {-1,  0,  1, -1, 1, -1, 0, 1, 0};
        __constant__ int c_yoff[9] = {-1, -1, -1,  0, 0,  1, 1, 1, 0};

        __device__ __forceinline__ int2 chooseRandomNeighbor(int x, int y, uint& randState, int count = 8)
        {
            int idx = nextRand(randState) % count;

            return make_int2(x + c_xoff[idx], y + c_yoff[idx]);
        }

        __device__ __forceinline__ uchar cvt(uchar val)
        {
            return val;
        }
        __device__ __forceinline__ uchar4 cvt(const uchar3& val)
        {
            return make_uchar4(val.x, val.y, val.z, 0);
        }
        __device__ __forceinline__ uchar4 cvt(const uchar4& val)
        {
            return val;
        }

        template <typename SrcT, typename SampleT>
        __global__ void init(const PtrStepSz<SrcT> frame, PtrStep<SampleT> samples, PtrStep<uint> randStates)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x >= frame.cols || y >= frame.rows)
                return;

            uint localState = randStates(y, x);

            for (int k = 0; k < c_nbSamples; ++k)
            {
                int2 np = chooseRandomNeighbor(x, y, localState, 9);

                np.x = ::max(0, ::min(np.x, frame.cols - 1));
                np.y = ::max(0, ::min(np.y, frame.rows - 1));

                SrcT pix = frame(np.y, np.x);

                samples(k * frame.rows + y, x) = cvt(pix);
            }

            randStates(y, x) = localState;
        }

        template <typename SrcT, typename SampleT>
        void init_caller(PtrStepSzb frame, PtrStepSzb samples, PtrStepSz<uint> randStates, hipStream_t stream)
        {
            dim3 block(32, 8);
            dim3 grid(divUp(frame.cols, block.x), divUp(frame.rows, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(init<SrcT), SampleT>, hipFuncCachePreferL1) );

            init<SrcT, SampleT><<<grid, block, 0, stream>>>((PtrStepSz<SrcT>) frame, (PtrStepSz<SampleT>) samples, randStates);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        void init_gpu(PtrStepSzb frame, int cn, PtrStepSzb samples, PtrStepSz<uint> randStates, hipStream_t stream)
        {
            typedef void (*func_t)(PtrStepSzb frame, PtrStepSzb samples, PtrStepSz<uint> randStates, hipStream_t stream);
            static const func_t funcs[] =
            {
                0, init_caller<uchar, uchar>, 0, init_caller<uchar3, uchar4>, init_caller<uchar4, uchar4>
            };

            funcs[cn](frame, samples, randStates, stream);
        }

        __device__ __forceinline__ int calcDist(uchar a, uchar b)
        {
            return ::abs(a - b);
        }
        __device__ __forceinline__ int calcDist(const uchar3& a, const uchar4& b)
        {
            return (::abs(a.x - b.x) + ::abs(a.y - b.y) + ::abs(a.z - b.z)) / 3;
        }
        __device__ __forceinline__ int calcDist(const uchar4& a, const uchar4& b)
        {
            return (::abs(a.x - b.x) + ::abs(a.y - b.y) + ::abs(a.z - b.z)) / 3;
        }

        template <typename SrcT, typename SampleT>
        __global__ void update(const PtrStepSz<SrcT> frame, PtrStepb fgmask, PtrStep<SampleT> samples, PtrStep<uint> randStates)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x >= frame.cols || y >= frame.rows)
                return;

            uint localState = randStates(y, x);

            SrcT imgPix = frame(y, x);

            // comparison with the model

            int count = 0;
            for (int k = 0; (count < c_reqMatches) && (k < c_nbSamples); ++k)
            {
                SampleT samplePix = samples(k * frame.rows + y, x);

                int distance = calcDist(imgPix, samplePix);

                if (distance < c_radius)
                    ++count;
            }

            // pixel classification according to reqMatches

            fgmask(y, x) = (uchar) (-(count < c_reqMatches));

            if (count >= c_reqMatches)
            {
                // the pixel belongs to the background

                // gets a random number between 0 and subsamplingFactor-1
                int randomNumber = nextRand(localState) % c_subsamplingFactor;

                // update of the current pixel model
                if (randomNumber == 0)
                {
                    // random subsampling

                    int k = nextRand(localState) % c_nbSamples;

                    samples(k * frame.rows + y, x) = cvt(imgPix);
                }

                // update of a neighboring pixel model
                randomNumber = nextRand(localState) % c_subsamplingFactor;

                if (randomNumber == 0)
                {
                    // random subsampling

                    // chooses a neighboring pixel randomly
                    int2 np = chooseRandomNeighbor(x, y, localState);

                    np.x = ::max(0, ::min(np.x, frame.cols - 1));
                    np.y = ::max(0, ::min(np.y, frame.rows - 1));

                    // chooses the value to be replaced randomly
                    int k = nextRand(localState) % c_nbSamples;

                    samples(k * frame.rows + np.y, np.x) = cvt(imgPix);
                }
            }

            randStates(y, x) = localState;
        }

        template <typename SrcT, typename SampleT>
        void update_caller(PtrStepSzb frame, PtrStepSzb fgmask, PtrStepSzb samples, PtrStepSz<uint> randStates, hipStream_t stream)
        {
            dim3 block(32, 8);
            dim3 grid(divUp(frame.cols, block.x), divUp(frame.rows, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(update<SrcT), SampleT>, hipFuncCachePreferL1) );

            update<SrcT, SampleT><<<grid, block, 0, stream>>>((PtrStepSz<SrcT>) frame, fgmask, (PtrStepSz<SampleT>) samples, randStates);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        void update_gpu(PtrStepSzb frame, int cn, PtrStepSzb fgmask, PtrStepSzb samples, PtrStepSz<uint> randStates, hipStream_t stream)
        {
            typedef void (*func_t)(PtrStepSzb frame, PtrStepSzb fgmask, PtrStepSzb samples, PtrStepSz<uint> randStates, hipStream_t stream);
            static const func_t funcs[] =
            {
                0, update_caller<uchar, uchar>, 0, update_caller<uchar3, uchar4>, update_caller<uchar4, uchar4>
            };

            funcs[cn](frame, fgmask, samples, randStates, stream);
        }
    }
}}}


#endif /* CUDA_DISABLER */